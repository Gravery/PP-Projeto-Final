#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//Merge realizado no device
__device__ void merge(int arr[], int l, int m, int r)
{
    int i, j, k;
    int n1 = m - l + 1;
    int n2 = r - m;

    int *L = (int*)malloc(n1 * sizeof(int));
    int *R = (int*)malloc(n2 * sizeof(int));

    for (i = 0; i < n1; i++)
        L[i] = arr[l + i];
    for (j = 0; j < n2; j++)
        R[j] = arr[m + 1 + j];

    i = 0;
    j = 0;
    k = l;
    while (i < n1 && j < n2) {
        if (L[i] <= R[j]) {
            arr[k] = L[i];
            i++;
        }
        else {
            arr[k] = R[j];
            j++;
        }
        k++;
    }

    while (i < n1) {
        arr[k] = L[i];
        i++;
        k++;
    }

    while (j < n2) {
        arr[k] = R[j];
        j++;
        k++;
    }

    free(L);
    free(R);
}

//Kernel do mergeSort que realiza chamadas recursivas
__global__ void mergeSort(int arr[], int l, int r)
{
    if (l < r) {
        int m = l + (r - l) / 2;
        mergeSort<<<1,1>>>(arr, l, m);
        mergeSort<<<1,1>>>(arr, m + 1, r);
        merge(arr, l, m, r);
    }
}

void printArray(int A[], int size)
{
    int i;
    for (i = 0; i < size; i++)
        printf("%d ", A[i]);
    printf("\n");
}

int main()
{
    int arr_size;
    fscanf(stdin, "%d", &arr_size);
    int *arr = (int*)malloc(arr_size * sizeof(int));

    for(int i = 0; i<arr_size; i++){
        scanf("%d", &arr[i]);
    }

    printf("Given array is \n");
    printArray(arr, arr_size);

    int *d_arr;
    hipMalloc((void**)&d_arr, arr_size * sizeof(int));
    hipMemcpy(d_arr, arr, arr_size * sizeof(int), hipMemcpyHostToDevice);

    mergeSort<<<1,1>>>(d_arr, 0, arr_size - 1);

    hipMemcpy(arr, d_arr, arr_size * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nSorted array is \n");
    printArray(arr, arr_size);

    free(arr);
    hipFree(d_arr);

    return 0;
}